#include "hip/hip_runtime.h"
/*
 *  gProximity Library.
 *  
 *  
 *  Copyright (C) 2010 University of North Carolina at Chapel Hill.
 *  All rights reserved.
 *  
 *  Permission to use, copy, modify, and distribute this software and its
 *  documentation for educational, research, and non-profit purposes, without
 *  fee, and without a written agreement is hereby granted, provided that the
 *  above copyright notice, this paragraph, and the following four paragraphs
 *  appear in all copies.
 *  
 *  Permission to incorporate this software into commercial products may be
 *  obtained by contacting the University of North Carolina at Chapel Hill.
 *  
 *  This software program and documentation are copyrighted by the University of
 *  North Carolina at Chapel Hill. The software program and documentation are
 *  supplied "as is", without any accompanying services from the University of
 *  North Carolina at Chapel Hill or the authors. The University of North
 *  Carolina at Chapel Hill and the authors do not warrant that the operation of
 *  the program will be uninterrupted or error-free. The end-user understands
 *  that the program was developed for research purposes and is advised not to
 *  rely exclusively on the program for any reason.
 *  
 *  IN NO EVENT SHALL THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL OR ITS
 *  EMPLOYEES OR THE AUTHORS BE LIABLE TO ANY PARTY FOR DIRECT, INDIRECT,
 *  SPECIAL, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS,
 *  ARISING OUT OF THE USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF THE
 *  UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL OR THE AUTHORS HAVE BEEN ADVISED
 *  OF THE POSSIBILITY OF SUCH DAMAGE.
 *  
 *  THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL AND THE AUTHORS SPECIFICALLY
 *  DISCLAIM ANY WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 *  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE AND ANY
 *  STATUTORY WARRANTY OF NON-INFRINGEMENT. THE SOFTWARE PROVIDED HEREUNDER IS
 *  ON AN "AS IS" BASIS, AND THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL AND
 *  THE AUTHORS HAVE NO OBLIGATIONS TO PROVIDE MAINTENANCE, SUPPORT, UPDATES,
 *  ENHANCEMENTS, OR MODIFICATIONS.
 *  
 *  Please send all BUG REPORTS to:
 *  
 *  geom@cs.unc.edu
 *  
 *  The authors may be contacted via:
 *  
 *  Christian Lauterbach, Qi Mo, Jia Pan and Dinesh Manocha
 *  Dept. of Computer Science
 *  Frederick P. Brooks Jr. Computer Science Bldg.
 *  3175 University of N.C.
 *  Chapel Hill, N.C. 27599-3175
 *  United States of America
 *  
 *  http://gamma.cs.unc.edu/GPUCOL/
 *  
 */
 
#include "bvh_kernels.h"

template <int nSplitThreads, int childNum>
__device__ __inline__ void storeSplit(const unsigned int thid, const unsigned int myOffset, const unsigned int numLeft, const AABBSplit &inSplit, float *localBoxes,
                                      AABBSplit *outputSplits, int *outputSplitMask, AABBNode *tree, const int *triIDs, AABBNode sharedNodeBuffer[2], const int nTrisPerLeaf)
{
        if(thid == 0)
        {
                AABBSplit outSplit;

                // Decide which child this is (templated, so the branch will be optimized away!)
                if(childNum == 0)    // LEFT child
                {
                        outSplit.left = inSplit.left;
                        outSplit.right = inSplit.left + numLeft - 1;
                        outSplit.myIndex = inSplit.nextIndex;
                        outSplit.nextIndex = inSplit.nextIndex + 2;
                }
                else   // RIGHT child
                {
                        outSplit.left = inSplit.left + numLeft;
                        outSplit.right = inSplit.right;
                        outSplit.myIndex = inSplit.nextIndex + 1;
                        outSplit.nextIndex = inSplit.nextIndex + 2 * numLeft;
                }

                int nPrimitives = outSplit.right - outSplit.left + 1;
                int isInnerNode;
                if(nTrisPerLeaf == 0)
                        isInnerNode = (nPrimitives > 1) ? nPrimitives : 0;
                else
                        isInnerNode = (nPrimitives > nTrisPerLeaf) ? nPrimitives : 0;
                outputSplitMask[myOffset*2 + childNum] = isInnerNode;
                outputSplits[myOffset*2 + childNum] = outSplit;

                // make leaf if necessary
                if(nTrisPerLeaf == 0)    // if just one tri, store the actual ID
                {
                        sharedNodeBuffer[childNum].leafTriID = (isInnerNode) ? 0 : ((triIDs[outSplit.left] << 2) | 3);
                }
                else   // otherwise, store interval in triangle index list
                {
                        sharedNodeBuffer[childNum].left  = (isInnerNode) ? 0 : ((outSplit.left << 2) | 3);
                        sharedNodeBuffer[childNum].right = outSplit.right;
                }

                // save to BVH
                /// TODO: could do optimized write by 8 threads for coalescing?
                sharedNodeBuffer[childNum].bbox.bb_min.x = localBoxes[0];
                sharedNodeBuffer[childNum].bbox.bb_min.y = localBoxes[nSplitThreads];
                sharedNodeBuffer[childNum].bbox.bb_min.z = localBoxes[2*nSplitThreads];
                sharedNodeBuffer[childNum].bbox.bb_max.x = localBoxes[3*nSplitThreads];
                sharedNodeBuffer[childNum].bbox.bb_max.y = localBoxes[4*nSplitThreads];
                sharedNodeBuffer[childNum].bbox.bb_max.z = localBoxes[5*nSplitThreads];
        }
}


template <int nSplitThreads, int childNum>
__device__ __inline__ void storeSplitLocal(const unsigned int thid, const unsigned int myOffset, const unsigned int numLeft, const AABBSplit &inSplit, float *localBoxes,
        WorkQueueItem *workQueue, int &workQueueIdx, const int *triIDs, const int *globalTriIDs, AABBNode &outputNode, const int nTrisPerLeaf)
{

        if(thid == 0)
        {
                AABBSplit outSplit;

                // Decide which child this is (templated, so the branch will be optimized away!)
                if(childNum == 0)    // LEFT child
                {
                        outSplit.left = inSplit.left;
                        outSplit.right = inSplit.left + numLeft - 1;
                        outSplit.myIndex = inSplit.nextIndex;
                        outSplit.nextIndex = inSplit.nextIndex + 2;
                }
                else   // RIGHT child
                {
                        outSplit.left = inSplit.left + numLeft;
                        outSplit.right = inSplit.right;
                        outSplit.myIndex = inSplit.nextIndex + 1;
                        outSplit.nextIndex = inSplit.nextIndex + 2 * numLeft;
                }

                int nPrimitives = outSplit.right - outSplit.left + 1;
                int isInnerNode;
                if(nTrisPerLeaf == 0)
                        isInnerNode = (nPrimitives > 1) ? nPrimitives : 0;
                else
                        isInnerNode = (nPrimitives > nTrisPerLeaf) ? nPrimitives : 0;

                // treat unused elements in localBoxes as shared
                ((int *)localBoxes)[31] = isInnerNode;

                //
                // save node
                //

                if(nTrisPerLeaf == 0)    // if just one tri, store the actual ID
                {
                        outputNode.leafTriID = (isInnerNode) ? 0 : ((globalTriIDs[triIDs[outSplit.left]] << 2) | 3);
                }
                else   // otherwise, store interval in triangle index list
                {
                        outputNode.left  = (isInnerNode) ? 0 : ((outSplit.left << 2) | 3);
                        outputNode.right = outSplit.right;
                }

                if(isInnerNode)
                {
                        workQueueIdx++;
                        workQueue[workQueueIdx].split = outSplit;
                }
        }

        __syncthreads();

        // save to BVH
        if(thid < 6)
                ((float *)&outputNode.bbox)[thid] = localBoxes[thid];

        //
        // if not a leaf node, write new item to work queue:
        //
        if(((int *)localBoxes)[31] && thid < 6)
                ((float *)&workQueue[workQueueIdx].nodeBB)[thid] = localBoxes[thid];

        __syncthreads();
}

__global__ void generateAABBsIndexed(float2 *boxes, int *triIDs, const GPUVertex *vertices, uint3* triangles, unsigned int *zCode,
                                     const unsigned int nTris, const unsigned int array_offset, const float3 sceneBB_min,
                                     const float3 sceneBB_max, const int nVerts)
{
	int triID = blockIdx.x * blockDim.x + threadIdx.x;
	float3 sceneDim = f3v_sub(sceneBB_max, sceneBB_min);
	//const float lim = float(UINT_MAX>>(32-10));
	const float lim = float(1 << LBVH_GRID_LEVELS);
	sceneDim.x = lim / sceneDim.x;
	sceneDim.y = lim / sceneDim.y;
	sceneDim.z = lim / sceneDim.z;
	
	// only when still within triangle bounds
	if(triID < nTris)
	{
		// start new AABB
		AABB newBB;
		uint3 triIndices = triangles[triID];
		float3 v = vertices[triIndices.x].v;
		newBB.bb_min = v;
		newBB.bb_max = v;
		
		v = vertices[triIndices.y].v;
		AABB_UPDATE(newBB, v);
		
		v = vertices[triIndices.z].v;
		AABB_UPDATE(newBB, v);
		
		// centroid of box
		float3 centroid = f3v_add(newBB.bb_min, newBB.bb_max);
		centroid = f3v_mul1(centroid, 0.5f);
		
		// quantize from scene BB
		centroid = f3v_sub(centroid, sceneBB_min);
		
		// quantized coordinate in scene BB
		int3 code3;
		code3.x = floor(centroid.x * sceneDim.x);
		code3.y = floor(centroid.y * sceneDim.y);
		code3.z = floor(centroid.z * sceneDim.z);
		
		// z code
		int code = 0;
		code3.x = code3.x << 2;
		code3.y = code3.y << 1;
		for(int j = 9; j >= 0; j--)
		{
			unsigned int triple = ((code3.x >> j) & 0x4)
			                      | ((code3.y >> j) & 0x2)
			                      | ((code3.z >> j) & 0x1);
			                      
			code = (code << 3) | triple;
		}
		
		if(zCode)
			zCode[triID] = code;
			
		// store AABB in 1-D array
		boxes[triID]		  = make_float2(newBB.bb_min.x, newBB.bb_max.x); // X
		unsigned int offset   = array_offset;
		boxes[offset + triID] = make_float2(newBB.bb_min.y, newBB.bb_max.y); // Y
		offset               += array_offset;
		boxes[offset + triID] = make_float2(newBB.bb_min.z, newBB.bb_max.z); // Z
		
		// store triID in linear array
		triIDs[triID] = triID;
	}
}

__global__ void AABBtoOBBbyLevel(AABBNode *treeIn, OBBNode *treeOut, unsigned int startOffset, unsigned int nNodes, const GPUVertex *vertices, const uint3 *triangles, const int vertexOffset)
{
	const int threadOffset = blockIdx.x * blockDim.x + threadIdx.x;
	const int myOffset = startOffset + threadOffset;
	
	if(threadOffset < nNodes)
	{
		// read in node child pointer
		unsigned int childPtr = treeIn[myOffset].left;
		treeOut[myOffset].left = childPtr;
		unsigned int childPtr2 = treeIn[myOffset].right;
		treeOut[myOffset].right = childPtr2;
		
		// if leaf, update from triangles
		if((childPtr & 3) == 3)
		{
			uint3 idx = triangles[childPtr >> 2];
			float3 v1 = vertices[idx.x].v;
			float3 v2 = vertices[idx.y].v;
			float3 v3 = vertices[idx.z].v;
			
			
			treeOut[myOffset].bbox.init(v1, v2, v3);
		}
		else   // otherwise, update from child nodes
		{
			unsigned int leftChild = myOffset + (childPtr >> 5);
			OBB bv1 = treeOut[leftChild].bbox;
			OBB bv2 = treeOut[leftChild+1].bbox;
			treeOut[myOffset].bbox.init(bv1, bv2);
		}
	}
}

__global__ void copyGPUVertex(GPUVertex* v1, float* v2, unsigned int nVertices, unsigned int nTargetVertices, bool useMin, unsigned int d)
{
	unsigned int threadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(threadId >= nTargetVertices)
		return;

	if(threadId >= nVertices)
	{
		v2[threadId] = useMin ? -HIP_NORM_HUGE_F : HIP_NORM_HUGE_F;
	}
	else
	{
		switch(d)
		{
		case 0:
			{
				v2[threadId] = v1[threadId].v.x;
			}
			break;
		case 1:
			{
				v2[threadId] = v1[threadId].v.y;
			}
			break;
		case 2:
			{
				v2[threadId] = v1[threadId].v.z;
			}
			break;
		}
	}
}
